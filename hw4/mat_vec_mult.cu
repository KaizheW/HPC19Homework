#include "hip/hip_runtime.h"
#include <algorithm>
#include <stdio.h>
#include <omp.h>
#include <string>

void VVMult_CPU(double* sum_ptr, const double* a, const double* b, long N){
  double sum = 0;
  #pragma omp parallel for schedule(static) reduction(+:sum)
  for (long i = 0; i < N; i++) sum += a[i]*b[i];
  *sum_ptr = sum;
}

void MVMult_CPU(double *C, double *A, double *B, long N) {
  #pragma omp parallel for
  for (long i = 0; i < N; i++) {
    for (long j = 0; j < N, j++) {
      double A_ij = A[i*N + j];
      double B_j = B[j];
      double C_i = C[i];
      C_i = C_i + A_ij * B_j;
      C[i] = C_i;
    }
  }
}

void Check_CUDA_Error(const char *message){
  hipError_t error = hipGetLastError();
  if(error!=hipSuccess) {
    fprintf(stderr,"ERROR: %s: %s\n", message, hipGetErrorString(error) );
    exit(-1);
  }
}

#define BLOCK_SIZE 1024

__global__ void reduction(double* sum, const double* a, long N){
  __shared__ double smem[BLOCK_SIZE];
  int idx = (blockIdx.x) * blockDim.x + threadIdx.x;

  if (idx < N) smem[threadIdx.x] = a[idx];
  else smem[threadIdx.x] = 0;

  __syncthreads();
  if (threadIdx.x < 512) smem[threadIdx.x] += smem[threadIdx.x + 512];
  __syncthreads();
  if (threadIdx.x < 256) smem[threadIdx.x] += smem[threadIdx.x + 256];
  __syncthreads();
  if (threadIdx.x < 128) smem[threadIdx.x] += smem[threadIdx.x + 128];
  __syncthreads();
  if (threadIdx.x <  64) smem[threadIdx.x] += smem[threadIdx.x +  64];
  __syncthreads();
  if (threadIdx.x <  32) {
    smem[threadIdx.x] += smem[threadIdx.x +  32];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +  16];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   8];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   4];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   2];
    __syncwarp();
    if (threadIdx.x == 0) sum[blockIdx.x] = smem[0] + smem[1];
  }
}

__global__ void product(double* sum, const double* A, const double* b, long N){
  __shared__ double smem[BLOCK_SIZE];
  int idx = (blockIdx.x) * blockDim.x + threadIdx.x;

  if (idx < N) smem[threadIdx.x] = A[idx]*b[idx];
  else smem[threadIdx.x] = 0;

  __syncthreads();
  if (threadIdx.x < 512) smem[threadIdx.x] += smem[threadIdx.x + 512];
  __syncthreads();
  if (threadIdx.x < 256) smem[threadIdx.x] += smem[threadIdx.x + 256];
  __syncthreads();
  if (threadIdx.x < 128) smem[threadIdx.x] += smem[threadIdx.x + 128];
  __syncthreads();
  if (threadIdx.x <  64) smem[threadIdx.x] += smem[threadIdx.x +  64];
  __syncthreads();
  if (threadIdx.x <  32) {
    smem[threadIdx.x] += smem[threadIdx.x +  32];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +  16];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   8];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   4];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   2];
    __syncwarp();
    if (threadIdx.x == 0) sum[blockIdx.x] = smem[0] + smem[1];
  }
}

int main() {
  long N = (1UL<<12);

  double *x, *A;
  hipHostMalloc((void**)&x, N * sizeof(double));
  hipHostMalloc((void**)&A, N*N*sizeof(double));
  #pragma omp parallel for schedule(static)
  for (long i = 0; i < N; i++) {
    x[i] = drand48();
  }
  for (long i = 0; i < N*N; i++) {
    A[i] = drand48();
  }

  double *sum_ref, *sum;
  for (long i = 0; i < N; i++) {
    sum_ref[i] = 0.0;
    sum[i] = 0.0;
  }
  double tt = omp_get_wtime();
  MVMult_CPU(&sum_ref, A, x, N);
  printf("CPU Bandwidth = %f GB/s\n", 1*N*sizeof(double) / (omp_get_wtime()-tt)/1e9);

  double *x_d, *A_d, *z_d;
  hipMalloc(&x_d, N*sizeof(double));
  hipMalloc(&A_d, N*N*sizeof(double));
  long N_work = 1;
  for (long i = (N+BLOCK_SIZE-1)/(BLOCK_SIZE); i > 1; i = (i+BLOCK_SIZE-1)/(BLOCK_SIZE)) N_work += i;
  hipMalloc(&z_d, N_work*sizeof(double)); // extra memory buffer for reduction across thread-blocks

  hipMemcpyAsync(x_d, x, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(A_d, A, N*N*sizeof(double), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  tt = omp_get_wtime();

  for (long i = 0; i < N; i++) {
    double* sum_d = z_d;
    long Nb = (N+BLOCK_SIZE-1)/(BLOCK_SIZE);
    product<<<Nb,BLOCK_SIZE>>>(sum_d, A_d, x_d, N);
    while (Nb > 1) {
      long N = Nb;
      Nb = (Nb+BLOCK_SIZE-1)/(BLOCK_SIZE);
      reduction<<<Nb,BLOCK_SIZE>>>(sum_d + N, sum_d, N);
      sum_d += N;
    }
    hipMemcpyAsync(&sum[i], sum_d, 1*sizeof(double), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
  }

  printf("GPU Bandwidth = %f GB/s\n", 1*N*sizeof(double) / (omp_get_wtime()-tt)/1e9);
  printf("Error = %f\n", fabs(sum-sum_ref));

  hipFree(x_d);
  hipFree(z_d);
  hipFree(A_d);
  hipHostFree(x);
  hipHostFree(A);

  return 0;
}
