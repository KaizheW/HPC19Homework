
#include <hip/hip_runtime.h>
#include <iostream>
#include <algorithm>
#include <stdio.h>
#include <omp.h>
#include <string>

void VVMult_CPU(double* sum_ptr, const double* a, const double* b, long N){
  double sum = 0;
  #pragma omp parallel for schedule(static) reduction(+:sum)
  for (long i = 0; i < N; i++) sum += a[i]*b[i];
  *sum_ptr = sum;
}

void MVMult_CPU(double* C, const double* A, const double* B, long N) {
  for (long i = 0; i < N; i++) {
    double sum = 0;
    #pragma omp parallel for schedule(static) reduction(+:sum)
    for (long j = 0; j < N; j++) {
      sum += A[i*N+j]*B[j];
    }
    C[i] = sum;
  }
}

#define BLOCK_SIZE 1024

__global__ void reduction(double* sum, const double* a, long N){
  __shared__ double smem[BLOCK_SIZE];
  int idx = (blockIdx.x) * blockDim.x + threadIdx.x;

  if (idx < N) smem[threadIdx.x] = a[idx];
  else smem[threadIdx.x] = 0;

  __syncthreads();
  if (threadIdx.x < 512) smem[threadIdx.x] += smem[threadIdx.x + 512];
  __syncthreads();
  if (threadIdx.x < 256) smem[threadIdx.x] += smem[threadIdx.x + 256];
  __syncthreads();
  if (threadIdx.x < 128) smem[threadIdx.x] += smem[threadIdx.x + 128];
  __syncthreads();
  if (threadIdx.x <  64) smem[threadIdx.x] += smem[threadIdx.x +  64];
  __syncthreads();
  if (threadIdx.x <  32) {
    smem[threadIdx.x] += smem[threadIdx.x +  32];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +  16];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   8];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   4];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   2];
    __syncwarp();
    if (threadIdx.x == 0) sum[blockIdx.x] = smem[0] + smem[1];
  }
}

__global__ void product(double* sum, const double* A, const double* b, long N){
  __shared__ double smem[BLOCK_SIZE];
  int idx = (blockIdx.x) * blockDim.x + threadIdx.x;

  if (idx < N) smem[threadIdx.x] = A[idx]*b[idx];
  else smem[threadIdx.x] = 0;

  __syncthreads();
  if (threadIdx.x < 512) smem[threadIdx.x] += smem[threadIdx.x + 512];
  __syncthreads();
  if (threadIdx.x < 256) smem[threadIdx.x] += smem[threadIdx.x + 256];
  __syncthreads();
  if (threadIdx.x < 128) smem[threadIdx.x] += smem[threadIdx.x + 128];
  __syncthreads();
  if (threadIdx.x <  64) smem[threadIdx.x] += smem[threadIdx.x +  64];
  __syncthreads();
  if (threadIdx.x <  32) {
    smem[threadIdx.x] += smem[threadIdx.x +  32];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +  16];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   8];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   4];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x +   2];
    __syncwarp();
    if (threadIdx.x == 0) sum[blockIdx.x] = smem[0] + smem[1];
  }
}

int main() {
  long N;
  int exp;
  std::cout << "N = 2^ " << std::endl;
  std::cin >> exp;
  N = (1UL<<exp);

  double *x, *A;
  hipHostMalloc((void**)&x, N * sizeof(double), hipHostMallocDefault);
  hipHostMalloc((void**)&A, N*N*sizeof(double), hipHostMallocDefault);
  #pragma omp parallel for schedule(static)
  for (long i = 0; i < N; i++) {
    x[i] = drand48();
  }
  for (long i = 0; i < N*N; i++) {
    A[i] = drand48();
  }

  double *sum_ref, *sum;
  hipHostMalloc((void**)&sum_ref, N * sizeof(double), hipHostMallocDefault);
  hipHostMalloc((void**)&sum, N * sizeof(double), hipHostMallocDefault);
/*  for (long i = 0; i < N; i++) {
    sum_ref[i] = 0.0;
    sum[i] = 0.0;
  }*/
  double tt = omp_get_wtime();
  MVMult_CPU(sum_ref, A, x, N);
  printf("CPU Bandwidth = %f GB/s\n", N*N*sizeof(double) / (omp_get_wtime()-tt)/1e9);

  double *x_d, *A_d, *z_d;
  hipMalloc(&x_d, N*sizeof(double));
  hipMalloc(&A_d, N*N*sizeof(double));
  long N_work = 1;
  for (long i = (N+BLOCK_SIZE-1)/(BLOCK_SIZE); i > 1; i = (i+BLOCK_SIZE-1)/(BLOCK_SIZE)) N_work += i;
  hipMalloc(&z_d, N_work*sizeof(double)); // extra memory buffer for reduction across thread-blocks

  hipMemcpyAsync(x_d, x, N*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpyAsync(A_d, A, N*N*sizeof(double), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  tt = omp_get_wtime();

  for (long i = 0; i < N; i++) {
    double* sum_d = z_d;
    long Nb = (N+BLOCK_SIZE-1)/(BLOCK_SIZE);
    product<<<Nb,BLOCK_SIZE>>>(sum_d, A_d+i*N, x_d, N);
    while (Nb > 1) {
      long Nx = Nb;
      Nb = (Nb+BLOCK_SIZE-1)/(BLOCK_SIZE);
      reduction<<<Nb,BLOCK_SIZE>>>(sum_d + Nx, sum_d, Nx);
      sum_d += Nx;
    }
    hipMemcpyAsync(&sum[i], sum_d, 1*sizeof(double), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
  }

  printf("GPU Bandwidth = %f GB/s\n", N*N*sizeof(double) / (omp_get_wtime()-tt)/1e9);
  double error = 0;
  #pragma omp parallel for reduction(+:error)
  for (long i = 0; i < N; i++) {
    error = error + fabs(sum[i] - sum_ref[i]);
  }
  printf("Error = %f\n", error);
  
  hipFree(x_d);
  hipFree(z_d);
  hipFree(A_d); 
  hipHostFree(x);
  hipHostFree(A);

  return 0;
}
